#include "hip/hip_runtime.h"
#include "cub_version.cuh"
#include "cuda_utils.cuh"

struct NotEqual
{
    int compare;
    HIPCUB_RUNTIME_FUNCTION __forceinline__ NotEqual(int compare)
        : compare(compare)
    {}
    HIPCUB_RUNTIME_FUNCTION __forceinline__ bool operator()(const int &a) const
    {
        return (a != compare);
    }
};

// Cub version
void fix_image(Image& to_fix)
{
    const int image_size = to_fix.width * to_fix.height;

    // #1 Compact

    constexpr int garbage_val = -27;

    int *d_in;
    hipMalloc(&d_in, to_fix.buffer.size() * sizeof(int));
    hipMemcpy(d_in, to_fix.buffer.data(), to_fix.buffer.size() * sizeof(int),
               hipMemcpyHostToDevice);

    int *d_image_buffer_fixed;
    hipMalloc(&d_image_buffer_fixed, image_size * sizeof(int));
    
    int *d_num_selected_out;
    hipMalloc(&d_num_selected_out, sizeof(int));
    
    NotEqual is_not_garbage(garbage_val);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_image_buffer_fixed,
                          d_num_selected_out, to_fix.buffer.size(), is_not_garbage);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_image_buffer_fixed,
                          d_num_selected_out, to_fix.buffer.size(), is_not_garbage);
    hipDeviceSynchronize();
    
    // num_selected_out is the number of elements that are not garbage, so it should be image_size
    int num_selected_out;
    hipMemcpy(&num_selected_out, d_num_selected_out, sizeof(int),
               hipMemcpyDeviceToHost);
    to_fix.buffer.resize(num_selected_out);
    hipMemcpy(to_fix.buffer.data(), d_image_buffer_fixed, num_selected_out * sizeof(int),
               hipMemcpyDeviceToHost);
    
    hipFree(d_in);
    hipFree(d_image_buffer_fixed);
    hipFree(d_num_selected_out);
    hipFree(d_temp_storage);
    
    // #2 Apply map to fix pixels

    for (int i = 0; i < image_size; ++i)
    {
        if (i % 4 == 0)
            to_fix.buffer[i] += 1;
        else if (i % 4 == 1)
            to_fix.buffer[i] -= 5;
        else if (i % 4 == 2)
            to_fix.buffer[i] += 3;
        else if (i % 4 == 3)
            to_fix.buffer[i] -= 8;
    }

    // #3 Histogram equalization

    // Histogram

    std::array<int, 256> histo;
    
    int *d_samples;
    hipMalloc(&d_samples, image_size * sizeof(int));
    hipMemcpy(d_samples, to_fix.buffer.data(), image_size * sizeof(int),
               hipMemcpyHostToDevice);

    int *d_histo;
    hipMalloc(&d_histo, 256 * sizeof(int));
    hipMemset(d_histo, 0, 256 * sizeof(int));
    
    void *d_temp_storage_histo = NULL;
    size_t temp_storage_bytes_histo = 0;
    hipcub::DeviceHistogram::HistogramEven(d_temp_storage_histo, temp_storage_bytes_histo,
                                        d_samples, d_histo, 256, 0, 255, image_size);
    hipMalloc(&d_temp_storage_histo, temp_storage_bytes_histo);
    hipcub::DeviceHistogram::HistogramEven(d_temp_storage_histo, temp_storage_bytes_histo,
                                        d_samples, d_histo, 256, 0, 255, image_size);
    hipDeviceSynchronize();

    hipFree(d_samples);
    hipFree(d_temp_storage_histo);
    
    // Computed d_histo is reused in the next step so no need to copy it back to host nor free it


    // Compute the inclusive sum scan of the histogram

    int *d_is_scan; // inclusive sum scan
    hipMalloc(&d_is_scan, 256 * sizeof(int));

    void *d_is_temp_storage = NULL;
    size_t is_temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_is_temp_storage, is_temp_storage_bytes, d_histo,
                                  d_is_scan, 256);
    hipMalloc(&d_is_temp_storage, is_temp_storage_bytes);
    hipcub::DeviceScan::InclusiveSum(d_is_temp_storage, is_temp_storage_bytes, d_histo,
                                  d_is_scan, 256);

    hipDeviceSynchronize();

    hipMemcpy(histo.data(), d_is_scan, 256 * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(d_is_scan);
    hipFree(d_is_temp_storage);
    
    hipFree(d_histo);

    // Find the first non-zero value in the cumulative histogram

    auto first_none_zero = std::find_if(histo.begin(), histo.end(), [](auto v) { return v != 0; });

    const int cdf_min = *first_none_zero;

    // Apply the map transformation of the histogram equalization

    std::transform(to_fix.buffer.data(), to_fix.buffer.data() + image_size, to_fix.buffer.data(),
        [image_size, cdf_min, &histo](int pixel)
            {
                return std::roundf(((histo[pixel] - cdf_min) / static_cast<float>(image_size - cdf_min)) * 255.0f);
            }
    );
}

// Cub main
int cub_main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[], Pipeline &pipeline)
{
    // -- Main loop containing image retring from pipeline and fixing

    const int nb_images = pipeline.images.size();
    // const int nb_images = 1;
    std::vector<Image> images(nb_images);

    // - One CPU thread is launched for each image
    std::cout << "Starting compute" << std::endl;

    #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        // You will need to copy images one by one on the GPU
        // You can store the images the way you want on the GPU
        // But you should treat the pipeline as a pipeline :
        // You *must not* copy all the images and only then do the computations
        // You must get the image from the pipeline as they arrive and launch computations right away
        // There are still ways to speeds this process of course (wait for last class)
        images[i] = pipeline.get_image(i);
        fix_image(images[i]);
    }

    std::cout << "Done with compute, starting stats" << std::endl;

    // -- All images are now fixed : compute stats (total then sort)

    // - First compute the total of each image

    #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        auto& image = images[i];
        const int image_size = image.width * image.height;
        //image.to_sort.total = std::reduce(image.buffer.cbegin(), image.buffer.cbegin() + image_size, 0);
        
        int *d_image;
        hipMalloc(&d_image, image_size * sizeof(int));
        hipMemcpy(d_image, image.buffer.data(), image_size * sizeof(int), hipMemcpyHostToDevice);
        
        int *d_reduce;
        hipMalloc(&d_reduce, image_size * sizeof(int));

        void     *d_temp_storage = NULL;
        size_t   temp_storage_bytes = 0; 
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                           d_image, d_reduce, image_size);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                           d_image, d_reduce, image_size);
        
        hipDeviceSynchronize();
        
        hipMemcpy(&image.to_sort.total, d_reduce, sizeof(int), hipMemcpyDeviceToHost);
        
        hipFree(d_image);
        hipFree(d_reduce);
        hipFree(d_temp_storage);
    }
    
    // - All totals are known, sort images accordingly (OPTIONAL)
    // Moving the actual images is too expensive, sort image indices instead
    // Copying to an id array and sort it instead
    
    auto image_totals = std::vector<int>(nb_images);
    std::transform(images.cbegin(), images.cend(), image_totals.begin(), [](const auto& image) { return image.to_sort.total; });
    auto image_indices = std::vector<int>(nb_images);
    std::transform(images.cbegin(), images.cend(), image_indices.begin(), [](const auto& image) { return image.to_sort.id; });

    // #pragma omp parallel for
    // for(int i = 0; i < nb_images; i++)
    // {
    //     image_indices[i] = to_sort[i].id;
    //     image_totals[i] = to_sort[i].total;
    // }

    int *d_keys_in;
    int *d_keys_out;
    int *d_values_in;
    int *d_values_out;

    hipMalloc(&d_keys_in, nb_images * sizeof(int));
    hipMalloc(&d_keys_out, nb_images * sizeof(int));
    hipMalloc(&d_values_in, nb_images * sizeof(int));
    hipMalloc(&d_values_out, nb_images * sizeof(int));

    hipMemcpy(d_keys_in, image_totals.data(), nb_images * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values_in, image_indices.data(), nb_images * sizeof(Image), hipMemcpyHostToDevice);
    
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                d_keys_in, d_keys_out,
                                d_values_in, d_values_out,
                                nb_images);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                d_keys_in, d_keys_out,
                                d_values_in, d_values_out,
                                nb_images);
    hipDeviceSynchronize();
    
    hipMemcpy(image_totals.data(), d_keys_out, nb_images * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(image_indices.data(), d_values_out, nb_images * sizeof(Image), hipMemcpyDeviceToHost);

    hipFree(d_keys_in);
    hipFree(d_keys_out);
    hipFree(d_values_in);
    hipFree(d_values_out);
    hipFree(d_temp_storage);
    
    // // - Print sorted images
    // std::cout << "Done with stats, starting output" << std::endl;
    // for (int i = 0; i < nb_images; ++i)
    // {
    //     // to align the output
    //     std::string s = image_indices[i] < 10 ? "0" : "";
    //     std::cout << "# Pre Sorting - Image " << s << image_indices[i] << " : " << image_totals[i] << std::endl;
    // }

    // TODO OPTIONAL : for you GPU version you can store it the way you want
    // But just like the CPU version, moving the actual images while sorting will be too slow
    using ToSort = Image::ToSort;
    std::vector<ToSort> to_sort(nb_images);
    std::generate(to_sort.begin(), to_sort.end(), [n = 0, images] () mutable
    {
        return images[n++].to_sort;
    });

    // TODO OPTIONAL : make it GPU compatible (aka faster)
    std::sort(to_sort.begin(), to_sort.end(), [](ToSort a, ToSort b) {
        return a.total < b.total;
    });


    // TODO : Test here that you have the same results
    // You can compare visually and should compare image vectors values and "total" values
    for (int i = 0; i < nb_images; ++i)
    {
        // If you did the sorting, check that the ids are in the same order
        assert(to_sort[i].id == image_indices[i]);

        std::string s = images[i].to_sort.id < 10 ? "0" : "";
        std::cout << "Image #" << s << images[i].to_sort.id << " total : " << images[i].to_sort.total << std::endl;
        std::ostringstream oss;
        oss << "Image#" << images[i].to_sort.id << ".pgm";
        std::string str = oss.str();
        images[i].write(str);
    }

    std::cout << "Done, the internet is safe now :)" << std::endl;

    return 0;
}