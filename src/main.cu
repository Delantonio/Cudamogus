#include "hip/hip_runtime.h"
#include "cub_version.cuh"
#include "baseline.hh"
#include "image.hh"
#include "pipeline.hh"
#include "fix_cpu.hh"
#include "cuda_utils.cuh"

#include <hipcub/hipcub.hpp>
#include <vector>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <filesystem>
#include <numeric>

template <typename T>
__global__ void kernel_shift(T* result, T* buffer, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= size)
    {
        return;
    }

    if (i == 0)
    {
        result[i] = 0;
    }
    else
    {
        result[i] = buffer[i - 1];
    }
}


template <typename T>
__global__
void kernel_inclusive_scan(T* buffer, T* scan_A, T* scan_P, int* blockstates, int* counter, int size) // scan_A + scan_P + blockstates are the same size (buffer_size / block_size)
{
    __shared__ int blockidx;
    
    if (threadIdx.x == 0)
    {
        blockidx = atomicAdd(counter, 1);
    }
    __syncthreads();
    
    int i = threadIdx.x + blockidx * blockDim.x;
    
    if (i >= size)
    {
        return;
    }

    // local scan
    for (int j = 1; j < blockDim.x; j *= 2)
    {
        int tmp = buffer[i - j];
        __syncthreads();

        if (i - blockidx * blockDim.x >= j)
            buffer[i] += tmp;
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        if (blockidx == 0)
        {
            atomicAdd(scan_P + blockidx, buffer[(blockidx+1) * blockDim.x - 1]);
            __threadfence_system();
            blockstates[blockidx] = 2;
        }
        else
        {
            atomicAdd(scan_A + blockidx, buffer[(blockidx+1) * blockDim.x - 1]);
            //__threadfence_system();
            blockstates[blockidx] = 1;
        }
    }
    __syncthreads();

    if (blockidx > 0)
    {
        //look back
        if (threadIdx.x == 0)
        {
            int idx = blockidx - 1;
            int state = atomicAdd(blockstates + idx, 0);
            //__threadfence_system();
            while (state != 2)
            {
                if (state == 1)
                {
                    int prev = atomicAdd(scan_A + idx, 0);
                    //__threadfence_system();
                    scan_P[blockidx] += prev;
                    idx--;
                }
                state = atomicAdd(blockstates + idx, 0);
                //__threadfence_system();
            }

            // prefix found
            int prevP = atomicAdd(scan_P + idx, 0);
            //__threadfence_system();
            int prevA = scan_A[idx];//atomicAdd(scan_A + idx, 0);
            //__threadfence_system();

            scan_P[blockidx] += prevP + prevA;
            blockstates[blockidx] = 2;
        }
        
        __syncthreads();
        buffer[i] += scan_P[blockidx];
    }
}

template <int BLOCK_SIZE>
__device__
void warp_reduce(int* sdata, int tid) {
    if (BLOCK_SIZE >= 64) {sdata[tid] += sdata[tid + 32]; __syncwarp(); }
    if (BLOCK_SIZE >= 32) {sdata[tid] += sdata[tid + 16]; __syncwarp(); }
    if (BLOCK_SIZE >= 16) {sdata[tid] += sdata[tid + 8]; __syncwarp(); }
    if (BLOCK_SIZE >= 8) {sdata[tid] += sdata[tid + 4]; __syncwarp(); }
    if (BLOCK_SIZE >= 4) {sdata[tid] += sdata[tid + 2]; __syncwarp(); }
    if (BLOCK_SIZE >= 2) {sdata[tid] += sdata[tid + 1]; __syncwarp(); }
}

template <typename T, int BLOCK_SIZE>
__global__
void kernel_reduce(const T* __restrict__ buffer, T* __restrict__ total, int size)
{
    extern __shared__ int sdata[];

    const unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    sdata[tid] = buffer[i] + buffer[i + blockDim.x];
    __syncthreads();

    if constexpr (BLOCK_SIZE >= 512) {
        if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
    }
    if constexpr (BLOCK_SIZE >= 256) {
        if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
    }
    if constexpr (BLOCK_SIZE >= 128) {
        if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
    }

    if (tid < 32)
        warp_reduce<BLOCK_SIZE>(sdata, tid);

    if (tid == 0) total[blockIdx.x] = sdata[0];
}

template <typename T>
__global__
void kernel_final_add(const T* __restrict__ buffer, T* __restrict__ total, int size)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < size)
        atomicAdd(&total[0], buffer[id]);
}

void reduce(CudaArray1D<int> buffer, CudaArray1D<int> total)
{
    constexpr int blocksize = 512;
    const int gridsize = (buffer.size_ + blocksize - 1) / (blocksize * 2);

    int *tmp;
    hipMalloc(&tmp, gridsize * sizeof(int));

	kernel_reduce<int, blocksize><<<gridsize, blocksize, blocksize * sizeof(int)>>>(buffer.data_, tmp, buffer.size_);
    kernel_final_add<int><<<gridsize / blocksize + 1, blocksize>>>(tmp, total.data_, gridsize);

    hipDeviceSynchronize();
    cudaCheckError();
}

/*__device__ __host__ void print_debug(int *image_data, int size) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i == 0)
    {
        for (int i = 0; i < size; i++)
            printf("DEBUG: image_data[%d] = %d\n", i, image_data[i]);
    }
    __syncthreads();
}*/

template<typename T>
__global__ void build_predicate(T *image_data, T* predicate, int image_size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= image_size)
        return;
    
    int garbage_value = -27;
    if (image_data[i] != garbage_value)
        predicate[i] = 1;
    
}

template<typename T>
__global__ void scatter_corresponding_adresses(T *image_data, T* image_data_backup, int* predicate, int image_size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= image_size)
        return;
    
    int garbage_value = -27;
    if (image_data_backup[i] != garbage_value)
        image_data[predicate[i]] = image_data_backup[i];
}

template<typename T>
__global__ void apply_map_to_pixels(T *image_data, int image_size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= image_size)
        return;
    
    if (i % 4 == 0)
    {
        image_data[i] += 1;
    }
    else if (i % 4 == 1)
    {
        image_data[i] -= 5;
    }
    else if (i % 4 == 2)
    {
        image_data[i] += 3;
    }
    else if (i % 4 == 3)
    {
        image_data[i] -= 8;
    }
    /*if (image_data[i] < 0 || image_data[i] > 255)
        printf("map bug image_data[%d] = %d\n", i, image_data[i]);*/
    //if (i < 20)
    //    printf("apply_map_pixel image_data[%d] = %d\n", i, image_data[i]);
    /*if (i % 4 == 0)
        atomicAdd(image_data + i, 1);//image_data[i] += 1;
    else if (i % 4 == 1)
        atomicAdd(image_data + i, -5);//image_data[i] -= 5;
    else if (i % 4 == 2)
        atomicAdd(image_data + i, 3);//image_data[i] += 3;
    else if (i % 4 == 3)
        atomicAdd(image_data + i, -8);//image_data[i] -= 8;*/
}

template<typename T>
__global__ void kernel_histogram(T *image_data, int *histogram, int image_size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= image_size)
        return;

    /*if (i == 0)
    {
        for (int j = 0; j < 20; j++)
        {
            printf("kernel_histogram image_data[%d] = %d\n", j, image_data[j]);
        }
    }
    
    __syncthreads();*/
    int image_value = image_data[i];
    /*if (image_value < 0 || image_value > 255)
        printf("histogram bug image_data[%d] = %d\n", i, image_value);*/
    atomicAdd(histogram + image_value, 1);
}

template<typename T>
__global__ void kernel_filter_zeros(T* histogram, int* predicate)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= 256)
        return;

    if (histogram[i] == 0)
    {
        predicate[i] = 0;
    }
    else
    {
        predicate[i] = 1;
    }
}

template<typename T>
__global__ void kernel_find_first_non_zero(T *histogram, int *summed_predicate, int* first_non_zero)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= 256)
        return;

    if (summed_predicate[i] == 1)
    {
        *first_non_zero = histogram[i];
    }
}

template<typename T>
__global__ void kernel_apply_map_transformation(T *result, T *image_data, int *histogram, int *first_non_zero, int image_size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= image_size)
        return;

    result[i] = std::roundf(((histogram[image_data[i]] - *first_non_zero) / static_cast<float>(image_size - *first_non_zero)) * 255.0f);
}

// Mano
void fix_image_gpu(int *image_data, int *image_data_cpu, const int image_size, const int buffer_size)
{
    constexpr int blocksize = 256;
    int nb_blocks = buffer_size / blocksize;
    nb_blocks++;

    printf("image_size = %d, buffer_size = %d\n", image_size, buffer_size);

    int *predicate;
    hipMalloc(&predicate, buffer_size * sizeof(int));
    hipMemset(predicate, 0, buffer_size * sizeof(int));

    build_predicate<int><<<nb_blocks, blocksize>>>(image_data, predicate, buffer_size);
    hipDeviceSynchronize();
    
    std::vector<int> predicate_cpu(buffer_size, 0);

    constexpr int garbage_val = -27;
    for (int i = 0; i < buffer_size; ++i)
        if (image_data_cpu[i] != garbage_val)
            predicate_cpu[i] = 1;

    int *predicate_gpu = (int*)malloc(sizeof(int) * buffer_size);
    hipMemcpy(predicate_gpu, predicate, sizeof(int) * buffer_size, hipMemcpyDeviceToHost);
    
    
    for (int i = 0; i < buffer_size; ++i)
    {
        if (predicate_cpu[i] != predicate_gpu[i])
        {
            printf("fail build predicate predicate[%d] = %d, predicate_gpu[%d] = %d\n", i, predicate_cpu[i], i, predicate_gpu[i]);
            exit(1);
        }
    }

    int* scan_A;
    hipMalloc(&scan_A, nb_blocks * sizeof(int));
    hipMemset(scan_A, 0, nb_blocks * sizeof(int));
    
    int* scan_P;
    hipMalloc(&scan_P, nb_blocks * sizeof(int));
    hipMemset(scan_P, 0, nb_blocks * sizeof(int));

    int* blockstates;
    hipMalloc(&blockstates, nb_blocks * sizeof(int));
    hipMemset(blockstates, 0, nb_blocks * sizeof(int)); // 0 = X; 1 == A; 2 == P

    int* counter;
    hipMalloc(&counter, sizeof(int));
    hipMemset(counter, 0, sizeof(int));

    // check if it is an exclusive scan
    kernel_inclusive_scan<int><<<nb_blocks, blocksize, sizeof(int)>>>(predicate, scan_A, scan_P, blockstates, counter, buffer_size);
    
    hipDeviceSynchronize();

    int* shifted_predicate;
    hipMalloc(&shifted_predicate, buffer_size * sizeof(int));
    hipMemset(shifted_predicate, 0, buffer_size * sizeof(int));

    kernel_shift<int><<<nb_blocks, blocksize>>>(shifted_predicate, predicate, buffer_size);

    hipDeviceSynchronize();

    std::exclusive_scan(predicate_cpu.begin(), predicate_cpu.end(), predicate_cpu.begin(), 0);
    
    hipMemcpy(predicate_gpu, shifted_predicate, sizeof(int) * buffer_size, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    
    for (int i = 0; i < buffer_size; ++i)
    {
        if (predicate_cpu[i] != predicate_gpu[i])
        {
            printf("scan fail predicate_cpu[%d] = %d, predicate_gpu[%d] = %d\n", i, predicate_cpu[i], i, predicate_gpu[i]);
            /*for (int j = i - 5; j < i + 5; j++)
            {
                printf("predicate_cpu[%d] = %d, predicate_gpu[%d] = %d\n", j, predicate_cpu[j], j, predicate_gpu[j]);
            }*/
            exit(1);
        }
    }

    int *image_data_copy;
    hipMalloc(&image_data_copy, buffer_size * sizeof(int));
    hipMemcpy(image_data_copy, image_data, buffer_size * sizeof(int), hipMemcpyDeviceToDevice);
    
    scatter_corresponding_adresses<int><<<nb_blocks, blocksize>>>(image_data, image_data_copy, shifted_predicate, buffer_size);
    hipDeviceSynchronize();
    
    int* image_data_gpu = (int*)malloc(sizeof(int) * buffer_size);
    hipMemcpy(image_data_gpu, image_data, sizeof(int) * buffer_size, hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < predicate_cpu.size(); ++i)
    {
        if (image_data_cpu[i] != garbage_val)
        {
            image_data_cpu[predicate_cpu[i]] = image_data_cpu[i];
        }
    }
    
    for (int i = 0; i < buffer_size; ++i)
    {
        if (image_data_cpu[i] != image_data_gpu[i])
        {
            printf("scatter fail image_cpu[%d] = %d, image_gpu[%d] = %d\n", i, image_data_cpu[i], i, image_data_gpu[i]);
            exit(1);
        }
    }

    apply_map_to_pixels<int><<<nb_blocks, blocksize>>>(image_data, image_size);
    hipDeviceSynchronize();
    hipMemcpy(image_data_gpu, image_data, sizeof(int) * image_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < image_size; ++i)
    {
        if (i % 4 == 0)
            image_data_cpu[i] += 1;
        else if (i % 4 == 1)
            image_data_cpu[i] -= 5;
        else if (i % 4 == 2)
            image_data_cpu[i] += 3;
        else if (i % 4 == 3)
            image_data_cpu[i] -= 8;
        if (image_data_cpu[i] < 0 || image_data_cpu[i] > 255)
            printf("cpu map bug image_data[%d] = %d\n", i, image_data_cpu[i]);
    }
    
    for (int i = 0; i < buffer_size; ++i)
    {
        if (image_data_cpu[i] != image_data_gpu[i])
        {
            printf("apply map fail image_cpu[%d] = %d, image_gpu[%d] = %d\n", i, image_data_cpu[i], i, image_data_gpu[i]);
            exit(1);
        }
    }

    // do histogram
    int* histogram;
    hipMalloc(&histogram, 256 * sizeof(int));
    hipMemset(histogram, 0, 256 * sizeof(int));

    kernel_histogram<int><<<nb_blocks, blocksize>>>(image_data, histogram, image_size);
    hipDeviceSynchronize();
    
    int *histogram_gpu = (int*)malloc(sizeof(int) * 256);
    hipMemcpy(histogram_gpu, histogram, sizeof(int) * 256, hipMemcpyDeviceToHost);

    std::array<int, 256> histo;
    histo.fill(0);
    for (int i = 0; i < image_size; ++i)
    {
        if (image_data_cpu[i] < 0 || image_data_cpu[i] > 255)
            printf("histogram bug image_data[%d] = %d\n", i, image_data_cpu[i]);
        ++histo[image_data_cpu[i]];
    }
    for (int i = 0; i < 256; ++i)
    {
        if (histo[i] != histogram_gpu[i])
        {
            printf("histogram fail histo_cpu[%d] = %d, histogram_gpu[%d] = %d\n", i, histo[i], i, histogram_gpu[i]);
            exit(1);
        }
    }

    // do inclusive scan
    hipMemset(scan_A, 0, nb_blocks * sizeof(int));
    hipMemset(scan_P, 0, nb_blocks * sizeof(int));
    hipMemset(blockstates, 0, nb_blocks * sizeof(int));
    hipMemset(counter, 0, sizeof(int));
    kernel_inclusive_scan<int><<<1, blocksize, sizeof(int)>>>(histogram, scan_A, scan_P, blockstates, counter, 256);
    hipDeviceSynchronize();
    
    hipMemcpy(histogram_gpu, histogram, sizeof(int) * 256, hipMemcpyDeviceToHost);

    std::inclusive_scan(histo.begin(), histo.end(), histo.begin());

    for (int i = 0; i < 256; ++i)
    {
        if (histo[i] != histogram_gpu[i])
        {
            printf("histogram inclusive scan fail histo_cpu[%d] = %d, histogram_gpu[%d] = %d\n", i, histo[i], i, histogram_gpu[i]);
            exit(1);
        }
    }
    
    //find first non zero
    int* first_non_zero;
    hipMalloc(&first_non_zero, sizeof(int));
    hipMemset(first_non_zero, 0, sizeof(int));

    int *predicate_find_first_non_zero;
    hipMalloc(&predicate_find_first_non_zero, 256 * sizeof(int));
    hipMemset(predicate_find_first_non_zero, 0, 256 * sizeof(int));

    kernel_filter_zeros<int><<<1, blocksize>>>(histogram, predicate_find_first_non_zero);
    hipDeviceSynchronize();

    hipMemset(scan_A, 0, nb_blocks * sizeof(int));
    hipMemset(scan_P, 0, nb_blocks * sizeof(int));
    hipMemset(blockstates, 0, nb_blocks * sizeof(int));
    hipMemset(counter, 0, sizeof(int));
    kernel_inclusive_scan<int><<<1, blocksize, sizeof(int)>>>(predicate_find_first_non_zero, scan_A, scan_P, blockstates, counter, 256);
    hipDeviceSynchronize();

    kernel_find_first_non_zero<int><<<1, blocksize>>>(histogram, predicate_find_first_non_zero, first_non_zero);
    hipDeviceSynchronize();
    
    auto first_none_zero_cpu = std::find_if(histo.begin(), histo.end(), [](auto v) { return v != 0; });

    const int cdf_min = *first_none_zero_cpu;

    int *first_non_zero_gpu = (int*)malloc(sizeof(int));
    hipMemcpy(first_non_zero_gpu, first_non_zero, sizeof(int), hipMemcpyDeviceToHost);

    if (cdf_min != *first_non_zero_gpu)
    {
        printf("find first non zero fail cdf_min = %d, first_non_zero_gpu = %d\n", cdf_min, *first_non_zero_gpu);
        exit(1);
    }

    //Apply map transformation of the histogram equalization
    hipMemcpy(image_data_copy, image_data, sizeof(int) * buffer_size, hipMemcpyDeviceToDevice);

    kernel_apply_map_transformation<int><<<nb_blocks, blocksize>>>(image_data, image_data_copy, histogram, first_non_zero, image_size);
    hipDeviceSynchronize();
    
    hipMemcpy(image_data_gpu, image_data, sizeof(int) * image_size, hipMemcpyDeviceToHost);
    
    std::transform(image_data_cpu, image_data_cpu + image_size, image_data_cpu,
        [image_size, cdf_min, &histo](int pixel)
            {
                return std::roundf(((histo[pixel] - cdf_min) / static_cast<float>(image_size - cdf_min)) * 255.0f);
            }
    );
    
    for (int i = 0; i < image_size; ++i)
    {
        if (image_data_cpu[i] != image_data_gpu[i])
        {
            printf("apply map transformation fail image_data_cpu[%d] = %d, image_data_gpu[%d] = %d\n", i, image_data_cpu[i], i, image_data_gpu[i]);
            exit(1);
        }
    }

    std::cout << "Done" << std::endl;
}

int gpu_main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[])
{
    // -- Pipeline initialization

    std::cout << "File loading..." << std::endl;

    // - Get file paths

    using recursive_directory_iterator = std::filesystem::recursive_directory_iterator;
    std::vector<std::string> filepaths;
    for (const auto& dir_entry : recursive_directory_iterator("../images"))
        filepaths.emplace_back(dir_entry.path().string());

    // - Init pipeline object

    Pipeline pipeline(filepaths);

    // -- Main loop containing image retring from pipeline and fixing

    const int nb_images = pipeline.images.size();
    std::vector<Image> images(nb_images);

    // - One CPU thread is launched for each image

    std::cout << "Done, starting compute" << std::endl;

    //#pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        images[i] = pipeline.get_image(i);
        std::cout << "Image " << i << " loaded " << images[i].to_sort.id << std::endl;
        int *image_data;
        int image_size = images[i].width * images[i].height;
        hipMalloc(&image_data, images[i].buffer.size() * sizeof(int));

        hipMemcpy(image_data, images[i].buffer.data(), images[i].buffer.size() * sizeof(int), hipMemcpyHostToDevice);
        fix_image_gpu(image_data, images[i].buffer.data(), image_size, images[i].buffer.size());

        std::cout << "Image " << i << " fixed " << images[i].to_sort.id << std::endl;

        hipMemcpy(images[i].buffer.data(), image_data, image_size * sizeof(int), hipMemcpyDeviceToHost);
        
        // for (int j = 50; j < image_size; j++)
        // {
        //     if (j == 80)
        //         break;
        //     std::cout << "DEBUG: image_data[" << j << "] = " << images[i].buffer[j] << std::endl;
        //     if (images[i].buffer[j] < 0 || images[i].buffer[j] > 255)
        //     {
        //         std::cout << "ERROR at "<< j << " : " << images[i].buffer[j] << " image_size: " << image_size << std::endl;
        //     }
        // }

        std::cout << "Image " << i << " copied " << images[i].to_sort.id << std::endl;
        
        std::ostringstream oss;
        oss << "ImageGPU#" << images[i].to_sort.id << ".pgm";
        std::string str = oss.str();
        images[i].write(str);
        std::cout << "Image " << i << " written" << std::endl;
    }

    return 0;
}


int main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[])
{
    {
        // std::cout << "cub version : " << std::endl;
        //cub_main(argc, argv);
    }
    
    if (argc > 1)
        cpu_main(argc, argv);
    else
        gpu_main(argc, argv);

    return 0;
}